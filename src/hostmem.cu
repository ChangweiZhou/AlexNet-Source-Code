#include <hostmem.cuh>

PinnedHostMem::PinnedHostMem() : _numBytes(0), _data(NULL) {

}

PinnedHostMem::~PinnedHostMem() {
    if (_numBytes > 0) {
        checkCudaErrors(hipHostFree(_data));
    }
}

void PinnedHostMem::resize(uint bytes) {
    if (_numBytes != bytes) {
        if (_numBytes > 0) {
            checkCudaErrors(hipHostFree(_data));
        }
        checkCudaErrors(hipHostAlloc(&_data, bytes, hipHostMallocPortable));
        _numBytes = bytes;
    }
}

void PinnedHostMem::copyFrom(void* src, uint bytes) {
    resize(bytes);
    checkCudaErrors(hipMemcpy(_data, src, bytes, hipMemcpyDefault));
}

void PinnedHostMem::copyTo(void* dst) {
    checkCudaErrors(hipMemcpy(dst, _data, _numBytes, hipMemcpyDefault));
}

void* PinnedHostMem::getData() {
    return _data;
}
